#include "hip/hip_runtime.h"
#include<iostream>
#include <thrust/sort.h>

using namespace std;

int main(int argc, char const *argv[]) {

	int numOfArrays = atoi(argv[1]);
	int maxElements = atoi(argv[2]);

	int N = numOfArrays*maxElements;
	float *data = new float[numOfArrays*maxElements];
	float *keys = new float[numOfArrays*maxElements];

	for(int i = 0; i < numOfArrays; i++){
		for(int j = 0; j < maxElements; j++){
			data [j+i*maxElements] = rand() % 501;
			keys [j+i*maxElements] = i;
		}
	}

	clock_t firstKrTime = clock();

	thrust::sort_by_key(data, data+N, keys);

	firstKrTime = clock()-firstKrTime;

	clock_t secondKrTime = clock();

	thrust::sort_by_key(keys, keys+N, data);

	secondKrTime = clock()-secondKrTime;

	cout<<(firstKrTime+secondKrTime)/double(CLOCKS_PER_SEC)*1000<<endl;


	free(data);
	free(keys);
	return 0;
}
