#include "hip/hip_runtime.h"
/*
compile using :
 nvcc -std=c++11 -arch=sm_35 -DnumOfArrays=<number of arrays> -DmaxElements=<maximum number of elements per array> GPU-ArraySort.cu -o out
*/


/*
Copyright (C) Muaaz Gul Awan and Fahad Saeed  
This program is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.
This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
*/



#include<iostream>
#include<vector>
#include<stdlib.h>
#include<time.h>
#include<math.h>
#include<ctime>
#include<algorithm>
#include<utility>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include<random>
using namespace std;

//#define numOfArrays 60000L
//#define maxElements 250
#define tempo 2
#define m 20
#define BUCKETS (maxElements/m)
#define sampleRate 10
#define SAMPLED (sampleRate*maxElements)/100
#define BLOCK_SIZE 1024



//data generation
template <typename mType>
struct dataArrays{
	vector<mType> dataList;
	int *prefixArray;
};


/* template <typename type> 
dataArrays<type> dataGen (int numOfArrays, int maxArraySize, int minArraySize){
	
   dataArrays<int> data;
   data.prefixArray = new int[numOfArrays+1]; //exclusive prefix scan
   const int range_from = 0;
   const unsigned int range_to = 30;//2147483647; //2^31 - 1
   random_device rand_dev;
   mt19937 generator(rand_dev());
   uniform_int_distribution<int> distr(range_from, range_to);
   int prefixSum = 0;
   srand(time(0));
	for( int i = 0; i < numOfArrays; i++){
	
		int size = rand()%(maxArraySize-minArraySize + 1) + minArraySize;
		data.prefixArray[i] = prefixSum;
		for(int j = prefixSum; j < prefixSum + size; j++){
			data.dataList.push_back(distr(generator));
		}
		prefixSum += size;
	}
	
	data.prefixArray[numOfArrays] = prefixSum;
	return data;
} */


//swap function for Insertion sort
template <class type>
 __device__ void  swapD (type &a, type &b)

{
    /* &a and &b are reference variables */
    type temp;
        temp=a;
	a=b;
        b=temp;
}

//insertion sort
template <class type>
 __device__ void insertionSort(type *input, int begin, int end){
        int i, j; //,tmp;
        for (i = begin+1; i < end; i++) {
            j = i;
            while (j > begin && input[j - 1] > input[j]) {
                  swapD(input[j], input[j-1]);
                   j--;
                 }//end of while loop
           }
        }




 int findArr(float input[], int size, int key){
          for(int i = 0; i < size; i++)
            {
               if(input[i] == key)
                  return 2;

            }

return 0;

}






__device__ void getMinMax(float input[], int beginPtr, int endPtr, float *ret){
          float min = input[beginPtr];
          float max = 0;
        // int *ret = new int[2];
          for(int i = beginPtr; i < endPtr; i++){
              if(min > input[i])
                  min = input[i];
              if (max < input[i])
                  max = input[i];     
            }

     ret[0] = min;
     ret[1] = max;
//return ret;

}

__device__ void getSplitters (float input[], float splitters[], int sample[], int beginPtr, int endPtr){
           __shared__ float mySamples[SAMPLED];
            float *ret = new float[2];
            for(int i = 0; i < SAMPLED; i++)
	   mySamples[i] = input[beginPtr+sample[i]];

	 insertionSort(mySamples, 0, SAMPLED);
       int splitterIndex = blockIdx.x*(BUCKETS+1)+1;
       int splittersSize=0;
	 for(int i = (SAMPLED)/(BUCKETS);splittersSize < BUCKETS-1; i +=SAMPLED/(BUCKETS)){
            splitters[splitterIndex] = mySamples[i];
            splitterIndex++;
            splittersSize++;
             }
          getMinMax(input, beginPtr, endPtr, ret);
           splitters[blockIdx.x*(BUCKETS+1)] = ret[0]-2;//to accodmodate the smallest
           splitters[blockIdx.x*(BUCKETS+1)+BUCKETS] = ret[1];
      
      delete [] ret;
}

__device__ void getBuckets2(float input[], float splitters[], int beginPtr, int endPtr, int bucketsSize[], float myInput[]){
      int id = threadIdx.x;
      int sizeOffset = blockIdx.x*BUCKETS+threadIdx.x;
      int bucketSizeOff = sizeOffset+1;
      float myBucket[maxElements];
      int indexSum=0;
      bucketsSize[bucketSizeOff] = 0;

     for(int i = 0; i < maxElements; i++){
         if(myInput[i] > splitters[id] && myInput[i] <= splitters[id+1]){
         myBucket[bucketsSize[bucketSizeOff]] = myInput[i];
         bucketsSize[bucketSizeOff]++;

}


     }
       
   __syncthreads();
   
         //prefix sum for bucket sizes of current array
         for(int j = 0; j < threadIdx.x; j++)
            indexSum += bucketsSize[blockIdx.x*BUCKETS+j+1];

         //writing back current buckt back to the input memory
	 for(int i = 0; i < bucketsSize[bucketSizeOff]; i++)
             input[indexSum+beginPtr+i] = myBucket[i];
       

}



__device__ void getBuckets(float input[], float splitters[], int beginPtr, int endPtr, int bucketsSize[]){
      int id = threadIdx.x;
      int sizeOffset = blockIdx.x*BUCKETS+threadIdx.x;
      int bucketSizeOff = sizeOffset+1;
      float myBucket[maxElements];
      int indexSum=0;
      bucketsSize[bucketSizeOff] = 0;

     for(int i = 0; i < maxElements; i++){
         if(input[beginPtr+i] > splitters[id] && input[beginPtr+i] <= splitters[id+1]){
         myBucket[bucketsSize[bucketSizeOff]] = input[beginPtr+i];
         bucketsSize[bucketSizeOff]++;

}


     }
       
   __syncthreads();
   
         //prefix sum for bucket sizes of current array
         for(int j = 0; j < threadIdx.x; j++)
            indexSum += bucketsSize[blockIdx.x*BUCKETS+j+1];

         //writing back current buckt back to the input memory
	 for(int i = 0; i < bucketsSize[bucketSizeOff]; i++)
             input[indexSum+beginPtr+i] = myBucket[i];
          

}
__device__ void bucketer(int input[], int bucketsSize[], int sample[], int beginPtr, int endPtr, int output[]){
          int id = blockIdx.x;
          const int toBeSampled = SAMPLED;
         const  int buckets = BUCKETS;
     
        __shared__ int splitters[buckets-1];
 

        //converting samples into unsorted-unselected-splitters

    for(int i = 0; i < toBeSampled; i ++)
        sample[i] = input[beginPtr+sample[i]];  
           insertionSort(sample, 0, toBeSampled);

         //taking splitters out
          int splittersSize=0;
	 for(int i = (toBeSampled)/(buckets);splittersSize < buckets-1; i +=toBeSampled/(buckets)){
             splitters[splittersSize] = sample[i];
            splittersSize++;
             }
        

          int sumBsize=0;
          int sIndex = 0;
          for(int i = id*BUCKETS; i < (id*BUCKETS+BUCKETS); i++){
              bucketsSize[i] = 0;
              for(int j = 0; j <maxElements ; j++){
              //for bucket 0
               if(sIndex == 0){
                if( input[beginPtr+j] <= splitters[0]){ 
		   output[beginPtr+sumBsize+bucketsSize[i]]=input[beginPtr+j];
                   bucketsSize[i]++;
                   } 
                    }
              //for last bucket
               else if(sIndex == buckets-1){
                    if( input[beginPtr+j] > splitters[splittersSize-1]){
                  
		   output[beginPtr+sumBsize+bucketsSize[i]] = input[beginPtr+j];
                   bucketsSize[i]++;
                       }
                  }
               else{
                    if( input[beginPtr+j] > splitters[sIndex-1] && input[beginPtr+j] <= splitters[sIndex]) {
		   output[beginPtr+sumBsize+bucketsSize[i]] = input[beginPtr+j];
                   bucketsSize[i]++;
                }

                }
             
	         }
                sumBsize += bucketsSize[i];
                sIndex++;
	      }



}


__global__ void splitterKer(float *data, float *splitters, int *mySample){
          if(blockIdx.x < numOfArrays){
             int id = blockIdx.x;
             int arrBegin = id*maxElements;
	     int arrEnd = arrBegin + maxElements;

	     __shared__ int sampleSh[SAMPLED];

	     for(int i = 0; i < SAMPLED; i++)
	        sampleSh[i] = mySample[i];

	     getSplitters(data, splitters, sampleSh, arrBegin, arrEnd);

          }
     }


__global__ void bucketEM2(float *data, int *bucketSizes, float *splittersGlob){
    if(blockIdx.x < numOfArrays){
        bucketSizes[0] = 0;
        int bid = blockIdx.x;
        int tid = threadIdx.x;
         int leftOvers = maxElements%BUCKETS;
         int jmpFac = maxElements/BUCKETS;
        int gArrayStart = bid*maxElements+tid*jmpFac;
        int gArrayEnd = (tid==(BUCKETS-1))?(gArrayStart + jmpFac+leftOvers):(gArrayStart + jmpFac);
        int lArrayStart = tid*jmpFac;
        __shared__ float myInput [maxElements];

        int arrBegin = bid*maxElements;
        int arrEnd = arrBegin + maxElements;
        int splitterIndexSt = blockIdx.x*(BUCKETS+1);
        int splitterIndexEd = splitterIndexSt + BUCKETS+1;
        __shared__ float splitters[BUCKETS+1];
//copy my array in shared memory in parallel
           for(int i=lArrayStart,j=gArrayStart;j<gArrayEnd;i++,j++){
                 myInput[i] = data[j];
           
        }
      __syncthreads(); 
        int j = 0;
        for(int i = splitterIndexSt; i < splitterIndexEd; i++){
           splitters[j] = splittersGlob[i];
           j++;
}
       
	getBuckets2(data, splitters, arrBegin, arrEnd, bucketSizes, myInput);

	}
}



__global__ void sortEM2(float *buckets, int *bucketSizes){
       if(blockIdx.x < numOfArrays && threadIdx.x < BUCKETS){
        int bid = blockIdx.x;
        int tid = threadIdx.x;
        int leftOvers = maxElements%BUCKETS;
        int jmpFac = maxElements/BUCKETS;
        int gArrayStart = bid*maxElements+tid*jmpFac;
         int gArrayEnd = (tid==(BUCKETS-1))?(gArrayStart + jmpFac+leftOvers):(gArrayStart + jmpFac);
        int lArrayStart = tid*jmpFac;

        __shared__ float myArray [maxElements];
        int indexSum = 0;
    

          for(int i=lArrayStart,j=gArrayStart;j<gArrayEnd;i++,j++){
                 myArray[i] = buckets[j];
           
        }
        __syncthreads();
          for(int j = 0; j < threadIdx.x; j++)
            indexSum += bucketSizes[blockIdx.x*BUCKETS+j+1];

 
          insertionSort(myArray, indexSum,indexSum+ bucketSizes[blockIdx.x*BUCKETS+threadIdx.x+1]);
          __syncthreads();
 

           for(int i=lArrayStart,j=gArrayStart;j<gArrayEnd;i++,j++){
                 buckets[j] = myArray[i];
           
        }
     __syncthreads();
}


}

__global__ void sortEM(int *buckets, int *prefixSum){

      if(blockIdx.x < numOfArrays && threadIdx.x < BUCKETS){
        int bid = blockIdx.x;
        int tid = threadIdx.x;

        int left =(tid)+bid*(BUCKETS);
        int right = (tid+1)+bid*(BUCKETS);
        
          insertionSort(buckets, prefixSum[left], prefixSum[right]); 
}
}



int main ()
{

   const int range_from = 0;
   const unsigned int range_to = 2147483647; //2^31 - 1
   random_device rand_dev;
   mt19937 generator(rand_dev());
   uniform_int_distribution<int> distr(range_from, range_to);
   size_t f,t;    
   int *d_bucketSizes , *h_bucketSizes;
   
   float *d_data, *h_buckets, *d_splitters;
    int numBlocks = ceil((float)(BUCKETS*numOfArrays+1)/(BLOCK_SIZE<<1));
    dim3 dimGrid(numBlocks, 1, 1);
    dim3 dimBlock(BLOCK_SIZE, 1, 1);
   float *h_data = new float[numOfArrays*maxElements];
   h_buckets = new float[numOfArrays*maxElements];
   h_bucketSizes = new int[BUCKETS*numOfArrays+1];
   int *prefix_Sum = new int[BUCKETS*numOfArrays+1];
   size_t size_heap, size_stack;
  int *h_sample = new int[SAMPLED];
  int *d_sample;
   hipSetDevice(0);
   hipMemGetInfo(&f, &t);

   //setting stack size limit
   hipDeviceSetLimit(hipLimitStackSize,10240);
   hipDeviceGetLimit(&size_heap, hipLimitMallocHeapSize);
   hipDeviceGetLimit(&size_stack, hipLimitStackSize);
   



//generating regular samples
       int max = maxElements;
      int  sam = SAMPLED;
      int stride = max/sam;
      int sampleVal = 0;
         for( int i = 0; i < SAMPLED; i++)
            {
               h_sample[i] = sampleVal;
               sampleVal += stride; 
    }


   // allocating device memory for data, sampled indices and bucket sizes
   hipMalloc((void**) &d_sample, SAMPLED*sizeof(float));
   hipMalloc((void**) &d_data, numOfArrays*maxElements*sizeof(float));
   hipMalloc((void**) &d_bucketSizes, numOfArrays*sizeof(int)*BUCKETS+sizeof(int));
     hipMalloc((void**) &d_splitters, (BUCKETS+1)*sizeof(float)*numOfArrays);
    srand(time(NULL));
     hipMemGetInfo(&f,&t);


//new data gens

for(int i = 0; i < numOfArrays; i++){
      for(int j = 0; j < maxElements; j++){
          h_data [j+i*maxElements] = distr(generator) ;
      }
   }

  

     //copy data and samples to GPU
     hipMemcpy(d_data, h_data, numOfArrays*maxElements*sizeof(float), hipMemcpyHostToDevice);
     hipMemcpy(d_sample, h_sample, SAMPLED*sizeof(float), hipMemcpyHostToDevice);

      
 
int firstKrTime = clock();
     splitterKer<<<numOfArrays,1>>>(d_data, d_splitters, d_sample);
hipDeviceSynchronize();
  firstKrTime = clock() - firstKrTime;


int secondKrTime = clock();        
     bucketEM2<<<numOfArrays,BUCKETS>>>(d_data, d_bucketSizes, d_splitters);
     hipDeviceSynchronize(); 
secondKrTime = clock()-secondKrTime;
     hipMemGetInfo(&f,&t);
	 
     //copying bucket sizes from first kernel back to cpu for prefix sum, to be replaced with prefix sum code
     hipMemcpy(h_bucketSizes, d_bucketSizes, sizeof(int)*(BUCKETS*numOfArrays+1), hipMemcpyDeviceToHost);
     //freeing the sample indices memory space and bucket sizes memory
     hipFree(d_sample);
     hipFree(d_splitters);
    
 


  

 
  

  
  


     int threeKrTime = clock();

    threeKrTime = clock() - threeKrTime;

    int fourKrTime = clock();
    sortEM2<<<numOfArrays, BUCKETS>>>(d_data, d_bucketSizes);
  
    hipDeviceSynchronize(); 

   fourKrTime = clock()-fourKrTime;


cout<<(firstKrTime+secondKrTime+threeKrTime+fourKrTime)/double(CLOCKS_PER_SEC)*1000<<endl;
   //copying the sorted data back
   hipMemcpy(h_buckets, d_data, numOfArrays*maxElements*sizeof(float), hipMemcpyDeviceToHost);



   

 
     //freeing the space for prefixSum and sorted data
     hipFree(d_data);
     hipFree(d_bucketSizes);
return 0;
}


